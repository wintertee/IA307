#include "hip/hip_runtime.h"
#include "fmatrix.cuh"
#include <assert.h>

#define THREADS_PER_BLOCK 1024

__global__
void fmatrix_multiplication_kernel(fmatrix A, float f, fmatrix B, fmatrix C) {
    // Each thread multiplies one row of B with one column of C
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int j = idx / A.rows;
    int i = idx % A.rows;
    if (i < A.rows && j < A.cols ){
        getfm(A,i,j) = 0.0;
        for (int k = 0; k < B.cols; ++k) {
          getfm(A,i,j) += f*getfm(B,i,k)*getfm(C,k,j);
        }
    }
}

/* Compute A = f*B*C */
void fmatrix_mult(fmatrix A, float f, fmatrix B, fmatrix C) {
    // First let's check for errors in the argument M.
    // This can help a LOT when debugging.
    // A,B,C need to have nonzero pointers etc.
    // fmatrix_assert(A);
    // fmatrix_assert(B);
    // fmatrix_assert(C);
    assert(A.rows == B.rows);
    assert(A.cols == C.cols);
    assert(B.cols == C.rows);

    // take one thread per element, and distribute
    // over as many blocks as necessary given
    // the hardware limit on the number of threads per block
    int threadsPerBlock = fmatrix_elements(A);
    int blocksPerGrid = 1;
    if (threadsPerBlock > THREADS_PER_BLOCK){
        blocksPerGrid = (threadsPerBlock-1)/THREADS_PER_BLOCK+1;
        threadsPerBlock = THREADS_PER_BLOCK;
    }
    fmatrix_multiplication_kernel<<< blocksPerGrid, threadsPerBlock >>>(A,f,B,C);
    // check for errors
    gpuErrchk( hipPeekAtLastError() );
    // wait for the kernel to finish
    device_synchronize();
}

