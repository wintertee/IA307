#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_stuff.cuh"

void device_synchronize(){
    gpuErrchk(hipDeviceSynchronize());
}
