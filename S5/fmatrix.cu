#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "cuda_stuff.cuh"
#include "fmatrix.cuh"

// for reading CSV files, we use some C++
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>

int fmatrix_elements(fmatrix mat) {
     return mat.cols*mat.rows;
}

int fmatrix_size(fmatrix mat) {
//    fmatrix_assert(mat);
     return fmatrix_elements(mat) * sizeof(float);
}

void fmatrix_assert(fmatrix mat) {
    assert(mat.data);
    assert(mat.cols);
    assert(mat.rows);
}

fmatrix fmatrix_create_on_host(int rows, int cols) {
    assert(cols>0);
    assert(rows>0);
    fmatrix mat;
    mat.cols = cols;
    mat.rows = rows;
    mat.data = (float*)malloc(fmatrix_size(mat));
    assert(mat.data);
    return mat;
}

fmatrix fmatrix_create_on_device(int rows, int cols) {
    assert(cols>0);
    assert(rows>0);
    fmatrix mat;
    mat.cols = cols;
    mat.rows = rows;
    gpuErrchk(
        hipMalloc((void **)&(mat.data), fmatrix_size(mat))
    );
    return mat;
}

void fmatrix_data_to_device(fmatrix mat_host, fmatrix mat_device) {
    fmatrix_assert(mat_host);
    fmatrix_assert(mat_device);
    assert(mat_host.cols==mat_device.cols);
    assert(mat_host.rows==mat_device.rows);
    gpuErrchk(
        hipMemcpy( mat_device.data, mat_host.data,
                   fmatrix_size(mat_host),
                   hipMemcpyHostToDevice
                   )
        );
}

void fmatrix_data_to_host(fmatrix mat_host, fmatrix mat_device) {
    fmatrix_assert(mat_host);
    fmatrix_assert(mat_device);
    assert(mat_host.cols==mat_device.cols);
    assert(mat_host.rows==mat_device.rows);
    gpuErrchk(
        hipMemcpy( mat_host.data, mat_device.data,
                   fmatrix_size(mat_device),
                   hipMemcpyDeviceToHost
                   )
        );
}

fmatrix fmatrix_copy_to_host(fmatrix mat_device) {
    fmatrix_assert(mat_device);
    fmatrix mat_host = fmatrix_create_on_host(mat_device.rows, mat_device.cols);
    fmatrix_data_to_host(mat_host,mat_device);
    return mat_host;
}

fmatrix fmatrix_copy_to_device(fmatrix mat_host) {
    fmatrix_assert(mat_host);
    fmatrix mat_device = fmatrix_create_on_device(mat_host.rows, mat_host.cols);
    fmatrix_data_to_device(mat_host,mat_device);
    return mat_device;
}

/** We could do it like this, but it would not set our pointer M.data to 0.
... fmatrix_free_on_host(M)
void fmatrix_free_on_host(fmatrix mat) {
    fmatrix_assert(mat);
  free(mat.data);
  mat.data = 0;
  mat.cols = 0;
  mat.rows = 0;
}
*/

void fmatrix_free_on_host(fmatrix* mat) {
    fmatrix_assert(*mat);
  free(mat->data);
  mat->data = 0;
  mat->cols = 0;
  mat->rows = 0;
}

void fmatrix_free_on_device(fmatrix* mat) {
    fmatrix_assert(*mat);
  gpuErrchk(hipFree(mat->data));
  mat->data = 0;
  mat->cols = 0;
  mat->rows = 0;
}

fmatrix fmatrix_subcolumns(fmatrix M, int a, int b) {
    fmatrix_assert(M);
    fmatrix A = {
        .data = &getfm(M,0,a),
        .cols = b-a,
        .rows = M.rows
    };
    fmatrix_assert(A);
    return A;
}


__host__
__device__
void fmatrix_print(fmatrix mat, int nb){
    if (nb<0 || nb > mat.rows) {
        nb = mat.rows;
    }
    printf("[\n");
    for (int i = 0 ; i < nb; i++){
      for (int j = 0 ; j<mat.cols; j++){
        printf("%f", getfm(mat,i,j));
        if (j+1<mat.cols) {
          printf(",\t");
        }
      }
      if (i+1<nb) {
        printf(";\n");
      }
    }
    if (nb < mat.rows) {
      printf("\n...\n");
    }
  printf("\n]\n");
}

void fmatrix_device_print(fmatrix mat, int nb){
   // allocate copy
   fmatrix tmp = fmatrix_copy_to_host(mat);
   fmatrix_print(tmp,nb);
   fmatrix_free_on_host(&tmp);
}

void fmatrix_device_to_csv(const char* filename, fmatrix mat) {
  // Open file
  FILE* fp = fopen(filename, "w");
  // allocate copy
  fmatrix tmp = fmatrix_copy_to_host(mat);
  for (int i = 0 ; i < tmp.rows; i++){
    for (int j = 0 ; j<tmp.cols; j++){
      // Note: %.15g gives 15 significant digits (full double precision)
      fprintf(fp,"%.15g", getfm(tmp,i,j));
      if (j+1<tmp.cols) {
        fprintf(fp,",");
      }
    }
    fprintf(fp,"\n");
  }
  fmatrix_free_on_host(&tmp);
  // Close file
  fclose(fp);
}

__global__
void fmatrix_create_random_on_device_kernel(fmatrix M) {
    // choose a seed (here: the same each launch)
    unsigned long seed = 0;
    int sequence = 0;
    // first, initialize the random numbers
    hiprandState state;
    hiprand_init(seed, sequence, 0, &state);
    for (int i = 0; i < fmatrix_elements(M); ++i) {
        // hiprand_uniform creates numbers between 0 and 1
        M.data[i] = (hiprand_uniform(&state)-0.5)*2.0;
    }
}

fmatrix fmatrix_create_random_on_device(int rows, int cols) {
    // Create an uninitialized matrix on the device
    fmatrix M = fmatrix_create_on_device(rows,cols);
    // Call a kernel with a single thread to fill the values
    fmatrix_create_random_on_device_kernel<<<1,1>>>(M);

    return M;
}

/* Count the number of rows and columns in a csv files (without headers) */
void count_elements_in_csv(const char* filename, int* rows, int* cols) {
  // Note: for the sake of convenience, we use some C++ functions here
  using namespace std;

  *rows = 0;
  *cols = 0;
  string row_as_string;
  string value;
  ifstream infile;
  infile.open(filename, ifstream::in);
	if (infile.is_open())
  {
    while (getline(infile, row_as_string, '\n')) {
				istringstream line_stream(row_as_string);
        int tempcols = 0;
        while (getline(line_stream, value, ',')) {
          ++tempcols;
        }
        if (tempcols > *cols) {
           *cols = tempcols;
        }
        ++(*rows);
			}
		infile.close();
	}
	else cout << "Cannot open file." << endl;
}

/** Read the data from a csv file into an fmatrix on the host.
 *  Careful: We assume that the matrix has the right dimensions!
 *  Use count_elements_in_csv(...) to get the dimensions if
 *  unknown.
 */
void fmatrix_fill_from_csv(fmatrix h_M,const char* filename) {
  // Note: for the sake of convenience, we use some C++ functions here
  using namespace std;
  string row_as_string;
  string value;
  ifstream infile;
  infile.open(filename, ifstream::in);
  int row = 0;
	if (infile.is_open())
  {
    while (getline(infile, row_as_string, '\n')) {
				istringstream line_stream(row_as_string);
        int col = 0;
        while (getline(line_stream, value, ',')) {
					getfm(h_M,row,col) = strtod(value.c_str(), NULL);
          ++col;
				}
        ++row;
			}
		infile.close();
	}
	else cout << "Cannot open file." << endl;
}

fmatrix fmatrix_device_from_csv(const char* filename) {
  // first read the file to count the number of elements
  int rows = 0;
  int cols = 0;
  count_elements_in_csv(filename,&rows,&cols);

  // allocate the matrix on the host
  fmatrix h_M = fmatrix_create_on_host(rows,cols);

  // read the data into the host matrix
  fmatrix_fill_from_csv(h_M,filename);

  // copy the matrix to the device
  fmatrix M = fmatrix_copy_to_device(h_M);

  // destroy the host matrix
  fmatrix_free_on_host(&h_M);

  return M;
}


