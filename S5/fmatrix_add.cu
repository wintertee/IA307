#include "hip/hip_runtime.h"
#include "fmatrix.cuh"
#include <assert.h>

#define THREADS_PER_BLOCK 1024
__global__
void fmatrix_add_kernel(fmatrix P,float a,fmatrix Y) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int j = idx / P.rows;
    int i = idx % P.rows;
    if (i < P.rows && j < P.cols ){
        getfm(P,i,j) += a*getfm(Y,i,j);
    }
}

/** Compute P = P + a*Y */
void fmatrix_add(fmatrix P,float a,fmatrix Y) {
    fmatrix_assert(P);
    fmatrix_assert(Y);
    assert(P.rows == Y.rows);
    assert(P.cols == Y.cols);
    int threadsPerBlock = fmatrix_elements(P);
    int blocksPerGrid = 1;
    if (threadsPerBlock > THREADS_PER_BLOCK){
        blocksPerGrid = (threadsPerBlock-1)/THREADS_PER_BLOCK+1;
        threadsPerBlock = THREADS_PER_BLOCK;
    }
    fmatrix_add_kernel<<< blocksPerGrid, threadsPerBlock >>>(P,a,Y);
    gpuErrchk( hipPeekAtLastError() );
}
