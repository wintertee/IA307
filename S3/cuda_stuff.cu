#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_stuff.cuh"

void device_synchronize()
{
    gpuErrchk(hipDeviceSynchronize());
}