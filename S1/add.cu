
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int a, int b, int *res)
{
    *res = a + b;
}

int main()
{
    printf("hello world\n");
    int res = 0;
    int *d_res = NULL;
    hipError_t err;

    // Launch add() kernel on GPU
    add<<<1, 1>>>(2, 2, d_res);
    err = hipPeekAtLastError();
    if (err != hipSuccess)
    {
        fprintf(stdout, "GPUassert: add launch failed with the error : %s \n", hipGetErrorString(err));
        exit(err);
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stdout, "GPUassert: add execution failed with the error : %s \n", hipGetErrorString(err));
        exit(err);
    }

    err = hipMemcpy(&res, d_res, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stdout, "GPUassert: hipMemcpy failed with the error : %s \n", hipGetErrorString(err));
        exit(err);
    }

    printf("2 + 2 = %d\n", res);

    return EXIT_SUCCESS;
}